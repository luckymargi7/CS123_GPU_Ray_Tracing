#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <float.h>

using namespace optix;

rtDeclareVariable(float4,  cylinder, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

__device__ bool withinConstraint(float3 eye, float3 dir, float t) {
    float3 interPt = eye + dir*t;

    float dist = interPt.x*interPt.x + interPt.z*interPt.z;

    return dist <= 0.25;
}

__device__ float instersectPlane(float3 eye, float3 dir, float3 point, float3 normal) {
    //Calculate t's
    float t = (dot(normal, point) - dot(eye, normal))/dot(dir, normal);
    
    return (withinConstraint(eye, dir, t)) ? t : -1;
        
}

template<bool use_robust_method>
__device__
void intersect_cylinder(void)
{
    float3 points[2];
    points[0] = float3(0, 0.5, 0);
    points[1] = float3(0,-0.5, 0);
    
    float3 normals[2];
    normals[0] = float3( 0, 1, 0);
    normals[1] = float3( 0,-1, 0);
    
    float3 eye = ray.origin;
    float3 dir = ray.direction;
    float final_t = FLT_MAX;
    int final_i = 0;

    for (int i = 0; i < 2; i++) {
        float3 p = points[i];
        float3 n = normals[i];
        float t = intersectPlane(eye, dir, p, n);

        if (rtPotentialIntersection(t) && t < final_t){
            final_t = t;
            final_i = i;
        }
    }
 
    //Calculate t's
    float a = dir.x*dir.x + dir.z*dir.z;
    float b = 2*eye.x*dir.x + 2*eye.z*dir.z;
    float c = eye.x*eye.x + eye.z*eye.z + 0.25;

    float d = b*b - 4*a*c;
    float t0 = t1 = -1;
    if (d >= 0) {
        t0 = (-1*b + sqrtf(d))/(2*a);
        t1 = (-1*b - sqrtf(d))/(2*a);
    }

    if (rtPotentialIntersection(t0) && t0 < final_t) {
        final_t = t0;
        final_i = -1;
    }
    
    if (rtPotentialIntersection(t1) && t1 < final_t) {
        final_t = t1;
        final_i = -1;
    }

    if (rtPotentialIntersection(final_t)){
        if (final_i != -1) {
            shading_normal = geometric_normal = normals[final_i];
        } else {
            float3 interPt = eye + dir(final_t);
            float3 normal = float3(interPt.x, 0, interPt.z);
            shading_normal = geometric_normal = normalize(normal);
        }

        rtReportIntersection(0);
    }
    
    
}


RT_PROGRAM void intersect(int primIdx)
{
  intersect_cylinder<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_cylinder<true>();
}


RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( cylinder );
  const float3 rad = make_float3( cylinder.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

