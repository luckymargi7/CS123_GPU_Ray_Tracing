#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <float.h>

using namespace optix;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

__device__ bool withinConstraint(float3 eye, float3 dir, float t) {
    float3 interPt = eye + dir*t;

    return fabs(interPt.x) <= 0.5 &&fabs(interPt.y) <= 0.5 && fabs(interPt.y) <= 0.5;
}

__device__ float instersectPlane(float3 eye, float3 dir, float3 point, float3 normal) {
    //Calculate t's
    float t = (dot(normal, point) - dot(eye, normal))/dot(dir, normal);
    
    return (withinConstraint(eye, dir, t)) ? t : -1;
        
}

RT_PROGRAM void cube_intersect(int)
{
    float3 points[6];
    points[0] = float3( 0.5, 0, 0);
    points[1] = float3(-0.5, 0, 0);
    points[2] = float3(0, 0.5, 0);
    points[3] = float3(0,-0.5, 0);
    points[4] = float3(0, 0, 0.5);
    points[5] = float3(0, 0,-0.5);
    
    float3 normals[6];
    normals[0] = float3( 1, 0, 0);
    normals[1] = float3(-1, 0, 0);
    normals[2] = float3( 0, 1, 0);
    normals[3] = float3( 0,-1, 0);
    normals[4] = float3( 0, 0, 1);
    normals[5] = float3( 0, 0,-1);
    
    float3 eye = ray.origin;
    float3 dir = ray.direction;
    float final_t = 0;
    int final_i = 0;

    for (int i = 0; i < 6; i++) {
        float3 p = points[i];
        float3 n = normals[i];
        float t = intersectPlane(eye, dir, p, n);

        if ((i == 0) || (rtPotentialIntersection(t) && t < final_t)){
            final_t = t;
            final_i = i;
        }
    }
 
    if (rtPotentialIntersection(final_t)) {
        shading_normal = geometric_normal = normals[final_i];
        rtReportIntersection(0);
    }

}

RT_PROGRAM void cube_bounds (int, float result[6])
{
    const float3 cubemin = make_float3(-.5,-.5,-.5);
    const float3 cubemax = make_float3(.5,.5,.5);
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(cubemin, cubemax);
}
