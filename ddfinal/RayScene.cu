#include "hip/hip_runtime.h"
/*
   I updated this to be specific to our needs as much as possible
   but basically everything is the same because this was a pretty
   much bare bones implementation of only the absolutely necessary
   things already.  (Much of it was taken from tutorial1.cu)
*/

#include "RayScene.h"

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );//regular normal
rtDeclareVariable(RayData_lighting, rd_lighting, rtPayload, );//ray struct defined in RayScene.h                                                                 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );//current t dist
rtDeclareVariable(uint2, currRay_index, rtLaunchIndex, );//TODO

//used to create the ray
rtDeclareVariable(unsigned int, lighting_ray_type, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );

//Define Camera
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, fail_color, , );
rtBuffer<uchar4, 2> output_buffer;

RT_PROGRAM void camera(){
    size_t2 screen = output_buffer.size();

    float2 d = make_float2(currRay_index) / make_float2(screen)*2.f-1.f;
    float3 ray_origin = eye;
    float3 ray_direction = normalized(d.x*U + d.y*V + W);

    optix::Ray ray(ray_origin, ray_direction, lighting_ray_type, scene_epsilon);
    RayData_lighting rd;
    rd.importance = 1.f;
    rd.depth = 0;

    rtTrace(top_object, ray, rd);

    output_buffer[currRay_index] = make_color( rd.result);
}

//Miss Method
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss(){
    rd_lighting.result = bg_color;
}

//Shading
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );

RT_PROGRAM void closest_hit_lighting(){
    float3 color = Ka * ambient_light_color;

    rd_lighting.result = color;
}

//Use failure color (if anything goes wrong)
RT_PROGRAM void exception(){
    output_buffer[currRay_index] = make_color(failure_color);
}




