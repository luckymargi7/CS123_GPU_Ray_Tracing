#include "hip/hip_runtime.h"
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4,  sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool use_robust_method>
__device__
void intersect_sphere(void)
{

    //Because of the transform groups, the ray should be transformed to hit the uniform sphere.
    float3 eye = ray.origin;
    float3 dir = ray.direction;
    
    float final_t = FLT_MAX; //Calculate t's
    float a = dot(dir, dir);
    float b = 2*eye.x*dir.x + 2*eye.y*dir.y + 2*eye.z*dir.z;
    float c = dot(eye, eye);

    float d = b*b - 4*a*c;
    float t0 = t1 = -1;
    if (d >= 0) {
        t0 = (-1*b + sqrtf(d))/(2*a);
        t1 = (-1*b - sqrtf(d))/(2*a);
    }
    
    if (rtPotentialIntersection(t0) && t0 < final_t) 
        final_t = t0;
    
    if (rtPotentialIntersection(t1) && t1 < final_t) 
        final_t = t1;
    
    if(rtPotentialIntersection(final_t))  {
        float3 interPt = eye + dir*t0;
        shading_normal = geometric_normal = normalize(interPt);
        rtReportIntersection(0);
    }

}

float3 normalize(float3 v) {
    return rsqrt(dot(v,v))*v;
}

RT_PROGRAM void intersect(int primIdx)
{
  intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_sphere<true>();
}


RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

