#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <float.h>

using namespace optix;

rtDeclareVariable(float4,  sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

template<bool use_robust_method>
__device__
void intersect_sphere(void)
{

    //Because of the transform groups, the ray should be transformed to hit the uniform sphere.
    float3 eye = ray.origin;
    float3 dir = ray.direction;
    
    float final_t = FLT_MAX; //Calculate t's
    float a = dot(dir, dir);
    float b = 2*eye.x*dir.x + 2*eye.y*dir.y + 2*eye.z*dir.z;
    float c = dot(eye, eye);

    float d = b*b - 4*a*c;
    float t0, t1;
    t0 = t1 = -1;
    if (d >= 0) {
        t0 = (-1*b + sqrtf(d))/(2*a);
        t1 = (-1*b - sqrtf(d))/(2*a);
    }
    
    if (rtPotentialIntersection(t0) && t0 < final_t) 
        final_t = t0;
    
    if (rtPotentialIntersection(t1) && t1 < final_t) 
        final_t = t1;
    
    if(rtPotentialIntersection(final_t))  {
        float3 interPt = eye + dir*t0;
        shading_normal = geometric_normal = normalize(interPt);
        rtReportIntersection(0);
    }

}

float3 normalize(float3 v) {
    return rsqrt(dot(v,v))*v;
}

RT_PROGRAM void intersect(int primIdx)
{
  intersect_sphere<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_sphere<true>();
}


RT_PROGRAM void bounds (int, float result[6])
{
    const float3 cubemin = make_float3(-.5,-.5,-.5);
    const float3 cubemax = make_float3(.5,.5,.5);
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(cubemin, cubemax);
}

