#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <float.h>

using namespace optix;

rtDeclareVariable(float4,  cone, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


__device__ bool withinConstraint(float3 eye, float3 dir, float t) {
    float3 interPt = eye + dir*t;

    float dist = interPt.x*interPt.x + interPt.z*interPt.z;

    return dist <= 0.25;
}

__device__ float instersectPlane(float3 eye, float3 dir, float3 point, float3 normal) {
    //Calculate t's
    float t = (dot(normal, point) - dot(eye, normal))/dot(dir, normal);
    
    return (withinConstraint(eye, dir, t)) ? t : -1;
        
}

template<bool use_robust_method>
__device__
void intersect_cone(void)
{
    float3 point = float3(0, -0.5, 0);
    float3 normal = float3( 0,-1, 0);
    
    float3 eye = ray.origin;
    float3 dir = ray.direction;
    float final_t = FLT_MAX;
    int final_i = 0;

    float t = intersectPlane(eye, dir, point, normal);

    if (rtPotentialIntersection(t) && t < final_t)
        final_t = t;
 
    //Calculate t's
    float a = dot(dir, dir);
    float b = 2*eye.x*dir.x + 2*eye.y*dir.y + 2*eye.z*dir.z;
    float c = dot(eye, eye);

    float d = b*b - 4*a*c;
    float t0 = t1 = -1;
    if (d >= 0) {
        t0 = (-1*b + sqrtf(d))/(2*a);
        t1 = (-1*b - sqrtf(d))/(2*a);
    }

    if (rtPotentialIntersection(t0) && t0 < final_t) {
        final_t = t0;
        final_i = -1;
    }
    
    if (rtPotentialIntersection(t1) && t1 < final_t) {
        final_t = t1;
        final_i = -1;
    }

    if (rtPotentialIntersection(final_t)){
        if (final_i != -1) {
            shading_normal = geometric_normal = normals[final_i];
        } else {
            float3 interPt = eye + dir(final_t);
            float normal_y = sqrtf(interPt.x*interPt.x + interPt.z*interPt.z)/2.0;
            float3 normal = float3(interPt.x, normal_y, interPt.z);
            shading_normal = geometric_normal = normalize(normal);
        }

        rtReportIntersection(0);
    }
}

float3 normalize(float3 v) {
    return rsqrt(dot(v,v))*v;
}

RT_PROGRAM void intersect(int primIdx)
{
  intersect_cone<false>();
}


RT_PROGRAM void robust_intersect(int primIdx)
{
  intersect_cone<true>();
}


RT_PROGRAM void bounds (int, float result[6])
{
  const float3 cen = make_float3( cone );
  const float3 rad = make_float3( cone.w );

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if( rad.x > 0.0f  && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

